#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstring>
#include <iostream>
#include <map>
#include "msbfs.hpp"
#include <cassert>
#include <ostream>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#define SEARCHES_PER_ENTRY 32
#define SEARCH_ENTRIES 32
#define SEARCHES_IN_WORKGROUP SEARCHES_PER_ENTRY * SEARCH_ENTRIES

struct SearchInfo {
  uint32_t iteration;
  uint32_t mask[SEARCH_ENTRIES];
  uint32_t jfq_length;
};

__global__ void set_first_bsak(uint32_t *bsak, uint32_t* src, uint32_t request_length) {
  for (uint32_t i = threadIdx.x; i < 32 && i + blockIdx.x * 32 < request_length; i += blockDim.x) {
    atomicOr(bsak + src[i + blockIdx.x * 32] * gridDim.x + blockIdx.x, 1u << i);
  }
}

__global__ void identify_step(uint32_t v_length, SearchInfo *info,
                              uint32_t *jfq, uint32_t *dst,
                              uint32_t *path_length, uint32_t *bsa,
                              uint32_t *bsak) {
  uint32_t c_mask = ~info->mask[threadIdx.x];
  uint32_t iteration = info->iteration;
  if (info[blockIdx.x].jfq_length == 0 && info[blockIdx.x].iteration > 0) {
    return;
  }
  info->jfq_length = 0;
  __syncthreads();

  for (uint32_t i = blockIdx.y * blockDim.y + threadIdx.y; i < v_length; i += gridDim.y * blockDim.y) {
    uint32_t diff = (bsa[i * blockDim.x + threadIdx.x] ^ bsak[i * blockDim.x + threadIdx.x]) & c_mask;
    if (__ballot_sync(~0, diff != 0) == 0) {
      continue;
    }

    bsak[i * blockDim.x + threadIdx.x] |= bsa[i * blockDim.x + threadIdx.x];
    uint32_t length = __popc(diff);
    for (uint32_t x = 0; x < length; x++) {
      uint32_t index = 31 - __clz(diff);
      if (dst[index + threadIdx.x * 32] == i) {
        path_length[index + threadIdx.x * 32] = iteration;
        c_mask &= ~(1 << index);
      }
      diff &= ~(1u << index);
    }

    if (threadIdx.x == 0) {
      jfq[atomicAdd(&info->jfq_length, 1u)] = i;
    }
  }
  atomicOr(&info->mask[threadIdx.x], ~c_mask);
  info->iteration = iteration + 1;
}
__global__ void expand_step(uint32_t v_length, uint32_t* v, uint32_t* e, SearchInfo *info, uint32_t *jfq,
                            uint32_t *bsa, uint32_t *bsak) {
  const uint32_t length = info[blockIdx.x].jfq_length;
  for (uint32_t i = blockIdx.y; i < length; i += gridDim.y) {
    const uint32_t source = jfq[i];
    const uint32_t val = bsa[source * blockDim.x + threadIdx.x];

    uint32_t start = v[source] + threadIdx.y;
    const uint32_t end = v[source + 1];
    for (; start < end; start += blockDim.y) {
      atomicOr(bsak + e[start] * blockDim.x + threadIdx.x, val);
    }
  }
}

std::vector<IterativeLengthResult> iterative_length(PathFindingRequest request,
                                                    CSR csr) {
  TimingInfo timing_info;
  return iterative_length(request, csr, timing_info);
}

std::vector<IterativeLengthResult> iterative_length(PathFindingRequest request,
                                                    CSR csr, TimingInfo &info) {
  hipSetDevice(0);

  uint64_t v_size = csr.v_length * sizeof(uint32_t);
  uint64_t e_size = csr.e_length * sizeof(uint32_t);

  uint32_t *src, *bsa, *bsak, *jfq, *v_buffer, *e_buffer, *dst, *path_lengths;
  uint32_t *host_result = new uint32_t[request.length];
  SearchInfo debug[1];
  SearchInfo *search_info;

  std::vector<IterativeLengthResult> results;
  results.reserve(request.length);

  hipMalloc(&v_buffer, v_size);
  hipMalloc(&e_buffer, e_size);

  hipMalloc(&bsa, v_size * SEARCH_ENTRIES);
  hipMalloc(&bsak, v_size * SEARCH_ENTRIES);
  hipMalloc(&jfq, v_size);

  hipMalloc(&dst, sizeof(uint32_t) * request.length);
  hipMalloc(&path_lengths, sizeof(uint32_t) * request.length);

  hipMalloc(&search_info, sizeof(SearchInfo));

  hipMalloc(&src, request.length * sizeof(uint32_t));

  hipMemcpy(v_buffer, csr.v, v_size, hipMemcpyHostToDevice);
  hipMemcpy(e_buffer, csr.e, e_size, hipMemcpyHostToDevice);
  hipMemcpy(src, request.src, request.length * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(dst, request.dst, request.length * sizeof(uint32_t), hipMemcpyHostToDevice);

  CSR cuda_csr = csr;
  cuda_csr.v = v_buffer;
  cuda_csr.e = e_buffer;

  for (size_t offset = 0; offset < request.length; offset += SEARCHES_IN_WORKGROUP) {
    // Clear BSA, BSAK, Destinations, Search Info.
    hipMemset(bsa, 0, v_size * SEARCH_ENTRIES);
    hipMemset(bsak, 0, v_size * SEARCH_ENTRIES);
    hipMemset(search_info, 0, sizeof(SearchInfo));
    // Setup BSAK
    set_first_bsak<<<SEARCH_ENTRIES, 32>>>(bsak, src + offset, request.length - offset);
    dim3 grid(1, 46 * 6, 1);
    dim3 block(SEARCH_ENTRIES, 4, 1);
    uint32_t jfq_lengths = 1;

    for (int iteration = 0; jfq_lengths > 0; iteration++) {
      if (iteration % 2 == 1) {
        identify_step<<<grid, dim3(SEARCH_ENTRIES, 4, 1)>>>(csr.v_length, search_info, jfq, dst + offset, path_lengths + offset, bsa, bsak);
        hipDeviceSynchronize();
        expand_step<<<grid, block>>>(cuda_csr.v_length, v_buffer, e_buffer, search_info, jfq, bsa, bsak);
      } else {
        identify_step<<<grid, dim3(SEARCH_ENTRIES, 4, 1)>>>(csr.v_length, search_info, jfq, dst + offset, path_lengths + offset, bsak, bsa);
        hipDeviceSynchronize();
        expand_step<<<grid, block>>>(cuda_csr.v_length, v_buffer, e_buffer, search_info, jfq, bsak, bsa);
      }

      hipDeviceSynchronize();
      if (iteration % 10 == 0) {
        hipMemcpy(debug, search_info, sizeof(SearchInfo), hipMemcpyDeviceToHost);
        jfq_lengths = debug[0].jfq_length;
      }
     }
  }
  hipMemcpy(host_result, path_lengths, request.length * sizeof(uint32_t), hipMemcpyDeviceToHost);
  for (size_t j = 0; j < request.length; j++) {
    if (host_result[j] == 0 && request.dst[j] != request.src[j]) {
      continue;
    }
    results.push_back({
      .src = request.src[j],
      .dst = request.dst[j],
      .length = host_result[j],
    });
  }

  hipFree(v_buffer);
  hipFree(e_buffer);

  hipFree(bsa);
  hipFree(bsak);
  hipFree(jfq);

  hipFree(dst);
  hipFree(path_lengths);

  hipFree(src);

  hipFree(search_info);


  return results;
}
